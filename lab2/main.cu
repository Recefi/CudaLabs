﻿
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <math.h>

__global__ void saxpyKernel(const int n, const float a, float* x, const int incX, float* y, const int incY) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
        y[i * incY] += a * x[i * incX];
}
__global__ void daxpyKernel(const int n, const double a, double* x, const int incX, double* y, const int incY) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
        y[i * incY] += a * x[i * incX];
}

void gpuSaxpy(const int n, const float a, float* x, const int incX, float* y, const int incY,
                                                            const int numBlocks, const int blockSize) {
    hipError_t cudaStatus;
    float* gpu_x = nullptr;
    float* gpu_y = nullptr;
    int xSize = 1 + (n - 1) * abs(incX);
    int ySize = 1 + (n - 1) * abs(incY);

    cudaStatus = hipMalloc((void**)&gpu_x, xSize * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc x failed!"); goto Error;
    }
    cudaStatus = hipMalloc((void**)&gpu_y, ySize * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc y failed!"); goto Error;
    }

    cudaStatus = hipMemcpy(gpu_x, x, xSize * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!"); goto Error;
    }
    cudaStatus = hipMemcpy(gpu_y, y, ySize * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!"); goto Error;
    }

    saxpyKernel<<<numBlocks, blockSize>>>(n, a, gpu_x, incX, gpu_y, incY);

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize error code: %d\n", cudaStatus);
        goto Error;
    }

    cudaStatus = hipMemcpy(y, gpu_y, ySize * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(gpu_x);
    hipFree(gpu_y);
}

void gpuDaxpy(const int n, const double a, double* x, const int incX, double* y, const int incY,
                                                                const int numBlocks, const int blockSize) {
    hipError_t cudaStatus;
    double* gpu_x = nullptr;
    double* gpu_y = nullptr;
    int xSize = 1 + (n - 1) * abs(incX);
    int ySize = 1 + (n - 1) * abs(incY);

    cudaStatus = hipMalloc((void**)&gpu_x, xSize * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc x failed!"); goto Error;
    }
    cudaStatus = hipMalloc((void**)&gpu_y, ySize * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc y failed!"); goto Error;
    }

    cudaStatus = hipMemcpy(gpu_x, x, xSize * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!"); goto Error;
    }
    cudaStatus = hipMemcpy(gpu_y, y, ySize * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!"); goto Error;
    }

    daxpyKernel<<<numBlocks, blockSize>>>(n, a, gpu_x, incX, gpu_y, incY);

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize error code: %d\n", cudaStatus);
        goto Error;
    }

    cudaStatus = hipMemcpy(y, gpu_y, ySize * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(gpu_x);
    hipFree(gpu_y);
}
