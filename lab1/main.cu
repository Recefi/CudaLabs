﻿
#include "hip/hip_runtime.h"


#include <stdio.h>

__global__ void printKernel() {
    int gIdx = blockIdx.x * blockDim.x + threadIdx.x;
    printf("I am from %d block, %d thread (global index: %d)\n", blockIdx.x, threadIdx.x, gIdx);
}
__global__ void calcKernel(int* a, unsigned int size) {
    int gIdx = blockIdx.x * blockDim.x + threadIdx.x;
    a[gIdx] = a[gIdx] + gIdx;
}

hipError_t printCalcGpu(int* a, unsigned int size) {
    int* gpu_a = nullptr;
    hipError_t cudaStatus;

    printKernel<<<2, 2>>>();

    cudaStatus = hipMalloc((void**)&gpu_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(gpu_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    calcKernel<<<2, 2>>>(gpu_a, size);

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize error code: %d\n", cudaStatus);
        goto Error;
    }

    cudaStatus = hipMemcpy(a, gpu_a, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(gpu_a);
    return cudaStatus;
}

int main() {
    const int size = 4;
    int a[size] = { 1, 2, 3, 4};

    printf("a = {%d,%d,%d,%d}\n", a[0], a[1], a[2], a[3]);

    hipError_t cudaStatus = printCalcGpu(a, size);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "printCalcGpu failed!");
        return 1;
    }

    printf("a = {%d,%d,%d,%d}\n", a[0], a[1], a[2], a[3]);

    return 0;
}
